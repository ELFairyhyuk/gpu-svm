#include "hip/hip_runtime.h"
/*
 * testTrainer.cpp
 *
 *  Created on: 31/10/2013
 *      Author: Zeyi Wen
 */

#include <sys/time.h>
#include "multiPredictor.h"
#include "trainClassifier.h"
#include "SVMCmdLineParser.h"
#include "classifierEvaluater.h"
#include "../svm-shared/Cache/cache.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"
#include "../SharedUtility/Timer.h"
#include "../SharedUtility/KeyValue.h"
#include "../SharedUtility/DataReader/LibsvmReaderSparse.h"

void trainSVM(SVMParam &param, string strTrainingFileName, int numFeature, SvmModel &model, bool evaluteTrainingError) {
    vector<vector<KeyValue> > v_v_Instance;
    vector<int> v_nLabel;

    int numInstance = 0;     //not used
    uint nNumofValue = 0;  //not used
    if(SVMCmdLineParser::numFeature > 0){
    	numFeature = SVMCmdLineParser::numFeature;
    }
    else
    	BaseLibSVMReader::GetDataInfo(strTrainingFileName, numFeature, numInstance, nNumofValue);
	LibSVMDataReader drHelper;
	drHelper.ReadLibSVMAsSparse(v_v_Instance, v_nLabel, strTrainingFileName, numFeature);
    SvmProblem problem(v_v_Instance, numFeature, v_nLabel);
//    problem = problem.getSubProblem(0,1);
    model.fit(problem, param);
    PRINT_TIME("training", trainingTimer)
    PRINT_TIME("working set selection",selectTimer)
    PRINT_TIME("pre-computation kernel",preComputeTimer)
    PRINT_TIME("iteration",iterationTimer)
    PRINT_TIME("g value updating",updateGTimer)
	model.saveLibModel(strTrainingFileName,problem);//save model in the same format as LIBSVM's
//    PRINT_TIME("2 instances selection",selectTimer)
//    PRINT_TIME("kernel calculation",calculateKernelTimer)
//    PRINT_TIME("alpha updating",updateAlphaTimer)
//    PRINT_TIME("init cache",initTimer)
    //evaluate training error
    if (evaluteTrainingError == true) {
        printf("Computing training accuracy...\n");
        //evaluate(model, v_v_Instance, v_nLabel, ClassifierEvaluater::trainingError);!!!!!!!!!!!!1not comment
	}
}

void evaluateSVMClassifier(SvmModel &model, string strTrainingFileName, int numFeature) {
    vector<vector<KeyValue> > v_v_Instance;
    vector<int> v_nLabel;

    int numInstance = 0;     //not used
    uint nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, numFeature, numInstance, nNumofValue);
	LibSVMDataReader drHelper;
	drHelper.ReadLibSVMAsSparse(v_v_Instance, v_nLabel, strTrainingFileName, numFeature);

    //evaluate testing error
    //evaluate(model, v_v_Instance, v_nLabel, ClassifierEvaluater::testingError);!!!!!!!!!!!!!!!!!!!1
}

/**
 * @brief: evaluate the svm model, given some labeled instances.
 */
void evaluate(SvmModel &model, vector<vector<KeyValue> > &v_v_Instance, vector<int> &v_nLabel,
			  vector<real> &classificationError, ofstream &ofs){
    int batchSize = 10000;

    //create a miss labeling matrix for measuring the sub-classifier errors.
    model.missLabellingMatrix = vector<vector<int> >(model.nrClass, vector<int>(model.nrClass, 0));
    bool bEvaluateSubClass = true; //choose whether to evaluate sub-classifiers
    if(model.nrClass == 2)  //absolutely not necessary to evaluate sub-classifers
        bEvaluateSubClass = false;

    MultiPredictor predictor(model, model.param);

	clock_t start, finish;
    start = clock();
    int begin = 0;
    vector<int> predictLabels;
    while (begin < v_v_Instance.size()) {
    	//get a subset of instances
    	int end = min(begin + batchSize, (int) v_v_Instance.size());
        vector<vector<KeyValue> > samples(v_v_Instance.begin() + begin,
                                          v_v_Instance.begin() + end);
        vector<int> vLabel(v_nLabel.begin() + begin, v_nLabel.begin() + end);
        if(bEvaluateSubClass == false)
        	vLabel.clear();
        //predict labels for the subset of instances
        vector<int> predictLabelPart = predictor.predict(samples, vLabel);
        predictLabels.insert(predictLabels.end(), predictLabelPart.begin(), predictLabelPart.end());
        begin += batchSize;
    }
    finish = clock();
    int numOfCorrect = 0;
    for (int i = 0; i < v_v_Instance.size(); ++i) {
        if (predictLabels[i] == v_nLabel[i])
            numOfCorrect++;
    }
    printf("classifier accuracy = %.2f%%(%d/%d)\n", numOfCorrect / (float) v_v_Instance.size() * 100,
           numOfCorrect, (int) v_v_Instance.size());
	ofs<<"binary training accuracy: "<<numOfCorrect/(float) v_v_Instance.size()*100<<" ("<<numOfCorrect<<"/"<<v_v_Instance.size()<<")"<<"\n";
    printf("prediction time elapsed: %.2fs\n", (float) (finish - start) / CLOCKS_PER_SEC);

    if(bEvaluateSubClass == true){
    	ClassifierEvaluater::evaluateSubClassifier(model.missLabellingMatrix, classificationError);
    }
}

float evaluateOVABinaryClassifier(vector<real>  &combDecValue, vector<vector<int> > &combPredictLabels, SvmModel &model, vector<vector<KeyValue> > &v_v_Instance, vector<int> &v_nLabel,
              vector<real> &classificationError){
    int batchSize = 10000;
    vector<real> decValue;
    //create a miss labeling matrix for measuring the sub-classifier errors.
    model.missLabellingMatrix = vector<vector<int> >(model.nrClass, vector<int>(model.nrClass, 0));
    bool bEvaluateSubClass = true; //choose whether to evaluate sub-classifiers
    if(model.nrClass == 2)  //absolutely not necessary to evaluate sub-classifers
        bEvaluateSubClass = false;

    MultiPredictor predictor(model, model.param);

    clock_t start, finish;
    start = clock();
    int begin = 0;
    vector<int> predictLabels;
    while (begin < v_v_Instance.size()) {
        //get a subset of instances
        int end = min(begin + batchSize, (int) v_v_Instance.size());
        vector<vector<KeyValue> > samples(v_v_Instance.begin() + begin,
                                          v_v_Instance.begin() + end);
        vector<int> vLabel(v_nLabel.begin() + begin, v_nLabel.begin() + end);
        if(bEvaluateSubClass == false)
            vLabel.clear();
        //predict labels for the subset of instances
       /* vector<int> predictLabelPart = predictor.predict(samples, vLabel);

        predictLabels.insert(predictLabels.end(), predictLabelPart.begin(), predictLabelPart.end());*/
        predictor.predictDecValue(decValue, samples);
        
		combDecValue.insert(combDecValue.end(), decValue.begin(), decValue.end());
        cout<<"before"<<endl;

        begin += batchSize;
    }
    finish = clock();
   /* //combine bianry predictLabels
    combPredictLabels.push_back(predictLabels);*/
        
    if(bEvaluateSubClass == true){
        ClassifierEvaluater::evaluateSubClassifier(model.missLabellingMatrix, classificationError);
    }
    return (float) (finish - start) / CLOCKS_PER_SEC;
}


void evaluateOVAVote(vector<vector<KeyValue> > &testInstance, vector<int> &testLabel, vector<vector<int> > &combPredictLabels, vector<int> &originalPositiveLabel, float testingTime){    //read test set
      //vote for class
    int manyClassIns=0;//#instance that belong to more than one classes
    int NoClassIns=0;//#instance that does't belong to any class
    int correctIns=0;
	int nrClass=originalPositiveLabel.size();
	//for(int i=0;i<10;i++)
    //	cout<<"testlabel********"<<combPredictLabels[0][i]<<endl;
	//cout<<"*******************"<<endl;
	//for(int i=0;i<10;i++)
    //	cout<<"testlabel********"<<combPredictLabels[1][i]<<endl;
	clock_t start,end;
	start=clock();
    for( int i=0;i<testInstance.size() ;i++){
        vector<int> vote(nrClass,0);
        int flag=0;
        int maxVote=0;
        for( int j=0;j<nrClass ;j++){
            if(combPredictLabels[j][i]==0)//if predictLabel=0 then instance belongs to the label 0 in jth bianrySVM
            {
			    vote[j]++;
                flag++;
                maxVote=j;
			}
        }
			
		if(i<10)
    		cout<<"flaglabel********"<<flag<<endl;
        if(flag==1){
            if(originalPositiveLabel[maxVote]==testLabel[i])
                correctIns++;
			cout<<"flag==1"<<endl;	
				}
        else if(flag>1){
            manyClassIns++;
			cout<<"many"<<endl;}
        else{
            NoClassIns++;
			cout<<"noclass"<<endl;
			}
		if(i<10)
    		cout<<"manyclasslabel********"<<manyClassIns<<endl;
    }
	end=clock();
	testingTime+=(float)(end-start)/CLOCKS_PER_SEC;
    printf("classifier accuracy = %.2f%%(%d/%d)\n", correctIns / (float) testInstance.size() * 100,
           correctIns, (int) testInstance.size() );
    printf("number of unclaasifiable instances in OVA is %.2f%%(%d/%d)\n", manyClassIns / (float) testInstance.size() * 100, manyClassIns, testInstance.size());
    printf("number of NoClass instances in OVA is %.2f%%(%d/%d)\n", NoClassIns / (float) testInstance.size() * 100, NoClassIns, testInstance.size() );
    printf("prediction time elapsed: %.2fs\n",testingTime);

    
}

void evaluateOVADecValue(vector<vector<KeyValue> > &testInstance, vector<int> &testLabel, vector<vector<real> > &combDecValue, vector<int > originalPositiveLabel, float testingTime, ofstream &ofs){    //read test set
      //vote for class
    int manyClassIns=0;//#instance that belong to more than one classes
    int noClassIns=0;//#instance that does't belong to any class
    int correctIns=0;
	int nrClass=originalPositiveLabel.size();

    //for(int i=0;i<10;i++)
    //  cout<<"testlabel********"<<combPredictLabels[0][i]<<endl;
    //cout<<"*******************"<<endl;
    //for(int i=0;i<10;i++)
    //  cout<<"testlabel********"<<combPredictLabels[1][i]<<endl;
    clock_t start,end;
    start=clock();
    for( int i=0;i<testInstance.size() ;i++){
        //vector<int> vote(nrClass,0);
        int flag=0;
        int max=0;
        for( int j=0;j<nrClass ;j++){
            if(combDecValue[j][i]>0&&combDecValue[j][i]>=combDecValue[max][i])//if predictLabel=0 then instance belongs to the label 0 in jth bianrySVM
            {
                //vote[j]++;
                flag++;
                max=j;
            }
        }
            
      
        if(flag>0){
            int manyClassflag=0;
            for(int j=0;j<nrClass ;j++){
                if(j!=max){
                    if(combDecValue[j][i]==combDecValue[max][i]){
                        manyClassIns++;
                        manyClassflag++;
                        break;//???
                    }
                }
                
            }
            if(manyClassflag==0){
                if (originalPositiveLabel[max]==testLabel[i])
                    correctIns++;
            }
        }
        else
            noClassIns++;
       
        // if(i<10)
        //     cout<<"manyclasslabel********"<<manyClassIns<<endl;
    }
    end=clock();
    testingTime+=(float)(end-start)/CLOCKS_PER_SEC;
    printf("test  accuracy = %.2f%%(%d/%d)\n", correctIns / (float) testInstance.size() * 100,
           correctIns, (int) testInstance.size() );
    ofs<<"test accuracy: "<<correctIns / (float) testInstance.size()*100<<" ("<<correctIns<<"/"<<testInstance.size()<<")"<<"\n";
    
	printf("number of unclaasifiable instances in OVA is %.2f%%(%d/%d)\n", manyClassIns / (float) testInstance.size() * 100, manyClassIns, testInstance.size());
    ofs<<"test: # of manyClass instance: "<<manyClassIns / (float) testInstance.size()*100<<" ("<<manyClassIns<<"/"<<testInstance.size()<<")"<<"\n";
    
	printf("number of NoClass instances in OVA is %.2f%%(%d/%d)\n", noClassIns / (float) testInstance.size() * 100, noClassIns, testInstance.size() );
    ofs<<"test: #noClass instance:  "<<noClassIns / (float) testInstance.size()*100<<" ("<<noClassIns<<"/"<<testInstance.size()<<")"<<"\n";
    printf("prediction time elapsed: %.2fs\n",testingTime);
	ofs<<"testing time: "<<testingTime<<"\n";

    
}

void trainOVASVM(SVMParam &param, string strTrainingFileName, int numFeature,  bool evaluteTrainingError, string strTestingFileName, ofstream &ofs) {
    //nrclass must >2
    //SvmModel model;//?????????????

    vector<vector<KeyValue> > v_v_Instance;
    vector<int> v_nLabel;

    int numInstance = 0;     //not used
    uint nNumofValue = 0;  //not used
    if(SVMCmdLineParser::numFeature > 0){
        numFeature = SVMCmdLineParser::numFeature;
    }
    else
        BaseLibSVMReader::GetDataInfo(strTrainingFileName, numFeature, numInstance, nNumofValue);
    LibSVMDataReader drHelper;
    drHelper.ReadLibSVMAsSparse(v_v_Instance, v_nLabel, strTrainingFileName, numFeature);
    //build problem of all classes
    SvmProblem problem(v_v_Instance, numFeature, v_nLabel);
cout<<"test"<<endl;
    int testNumInstance = 0;     //not used
    uint testNumofValue = 0;
    vector<vector<KeyValue> > testInstance;
    vector<int> testLabel;
    BaseLibSVMReader::GetDataInfo(strTestingFileName, numFeature, testNumInstance, testNumofValue);
    drHelper.ReadLibSVMAsSparse(testInstance, testLabel, strTestingFileName, numFeature);
    
	int nrClass=problem.getNumOfClasses();
    //vector<SvmModel> combModel(nrClass);
    vector<vector<int> > combPredictLabels;//combine k binary predictLaebl
    vector<vector<real> > combDecValue(nrClass);
    vector<vector<int> > combTrainPredictLabels;//combine k binary predictLaebl
    vector<int> originalPositiveLabel(nrClass);
    float testingTime=0;
    float allTrainingTime=0;
    float avgTrainingTime=0;
    for(int i=0;i<nrClass;i++)
        originalPositiveLabel[i]=v_nLabel[problem.perm[problem.start[i]]];
    
    //train and predict bianry svm
    for(int i=0;i<nrClass;i++){
        SvmModel model;
        //reassign the 0 and 1 label to instances.
        for(int m=0;m<problem.count[i];m++)
            v_nLabel[problem.perm[problem.start[i] + m]]=0;//0 denotes the positive class
        for(int n=0;n<nrClass;n++){
            if(n!=i){
                for(int l=0;l<problem.count[n];l++)
                    v_nLabel[problem.perm[problem.start[n] + l]]=1;
            }
        }
        //for class i=0, in training ing phase, class i=0 will be +1,
        //for other classes (i!=0), in training ing phase, class i will be -1
        
        //use instance with label 0 and 1 to build the problem
        SvmProblem binaryProblem(v_v_Instance, numFeature, v_nLabel);
        //problem.label=model.label  label[0]=the label of the first instance.

        model.fit(binaryProblem, param);//resize nrclass!!!!!solve->getsubproblem!!!!
        PRINT_TIME("training", trainingTimer)
        PRINT_TIME("working set selection",selectTimer)
        PRINT_TIME("pre-computation kernel",preComputeTimer)
        PRINT_TIME("iteration",iterationTimer)
        PRINT_TIME("g value updating",updateGTimer)
        model.saveLibModel(strTrainingFileName,problem);//save model in the same format as LIBSVM's
		allTrainingTime+=trainingTimer.getTotalTime();
		avgTrainingTime+=trainingTimer.getAverageTime();
//    PRINT_TIME("2 instances selection",selectTimer)
//    PRINT_TIME("kernel calculation",calculateKernelTimer)
//    PRINT_TIME("alpha updating",updateAlphaTimer)
//    PRINT_TIME("init cache",initTimer)
    //evaluate training error
        if (evaluteTrainingError == true) {
            printf("Computing training accuracy...\n");
            evaluate(model, v_v_Instance, v_nLabel, ClassifierEvaluater::trainingError, ofs);
        }

        cout << "start evaluation..." << endl;
        testingTime+= evaluateOVABinaryClassifier(combDecValue[i], combPredictLabels, model, testInstance, testLabel, ClassifierEvaluater::testingError);
  //      evaluateOVABinaryClassifier(combTrainPredictLabels, model, v_v_Instance, v_nLabel, ClassifierEvaluater::testingError);
   
    }
    ofs<<"total training time "<<allTrainingTime<<" avg time"<<avgTrainingTime<<"\n";
	cout<<"all evaluation"<<endl;
    evaluateOVADecValue(testInstance, testLabel, combDecValue, originalPositiveLabel, testingTime, ofs);    //read test set

	//evaluateOVAVote(testInstance, testLabel, combPredictLabels, originalPositiveLabel, testingTime);
//	evaluateOVA(v_v_Instance, v_nLabel, combTrainPredictLabels, originalPositiveLabel, testingTime);
}
