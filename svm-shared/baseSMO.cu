#include "hip/hip_runtime.h"
/*
 * baseSMO.cu
 *  @brief: definition of some sharable functions of smo solver
 *  Created on: 24 Dec 2016
 *      Author: Zeyi Wen
 */

#include "baseSMO.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "smoGPUHelper.h"
#include "../SharedUtility/Timer.h"

/**
 * @brief: initialise some variables of smo solver
 */
void BaseSMO::InitSolver(int nNumofTrainingIns)
{
	alpha = vector<float_point>(nNumofTrainingIns, 0);

    configureCudaKernel(nNumofTrainingIns);
	//allocate device memory for min/max search
	checkCudaErrors(hipMalloc((void**)&devBlockMin, sizeof(float_point) * numOfBlock));
	checkCudaErrors(hipMalloc((void**)&devBlockMinGlobalKey, sizeof(int) * numOfBlock));
	//for getting maximum low G value
	checkCudaErrors(hipMalloc((void**)&devBlockMinYiGValue, sizeof(float_point) * numOfBlock));
	checkCudaErrors(hipMalloc((void**)&devMinValue, sizeof(float_point)));
	checkCudaErrors(hipMalloc((void**)&devMinKey, sizeof(int)));

	checkCudaErrors(hipHostMalloc((void **) &hostBuffer, sizeof(float_point) * 5));
	checkCudaErrors(hipMalloc((void**)&devBuffer, sizeof(float_point) * 5));//only need 4 float_points

	//diagonal is frequently used in training.
	hessianDiag = new float_point[nNumofTrainingIns];
    checkCudaErrors(hipMalloc((void **) &devHessianDiag, sizeof(float_point) * nNumofTrainingIns));
}

/**
 * @brief: release solver memory
 */
void BaseSMO::DeInitSolver()
{
    checkCudaErrors(hipFree(devBlockMin));
    checkCudaErrors(hipFree(devBlockMinGlobalKey));
    checkCudaErrors(hipFree(devBlockMinYiGValue));
    checkCudaErrors(hipFree(devMinValue));
    checkCudaErrors(hipFree(devMinKey));
    checkCudaErrors(hipFree(devBuffer));
    checkCudaErrors(hipHostFree(hostBuffer));
    checkCudaErrors(hipFree(devHessianDiag));
    delete[] hessianDiag;
}

/**
 * @brief: select the first instance in SMO
 */
void BaseSMO::SelectFirst(int numTrainingInstance, float_point CforPositive)
{
    TIMER_START(selectTimer)
	GetBlockMinYiGValue<<<gridSize, BLOCK_SIZE>>>(devYiGValue, devAlpha, devLabel, CforPositive,
														   numTrainingInstance, devBlockMin, devBlockMinGlobalKey);
	//global reducer
	GetGlobalMin<<<1, BLOCK_SIZE>>>(devBlockMin, devBlockMinGlobalKey, numOfBlock, devYiGValue, NULL, devBuffer);

	//copy result back to host
	hipMemcpy(hostBuffer, devBuffer, sizeof(float_point) * 2, hipMemcpyDeviceToHost);
	IdofInstanceOne = (int)hostBuffer[0];
    TIMER_STOP(selectTimer)

	devHessianInstanceRow1 = ObtainRow(numTrainingInstance);
}

/**
 * @breif: select the second instance in SMO
 */
void BaseSMO::SelectSecond(int numTrainingInstance, float_point CforNegative)
{
    TIMER_START(selectTimer)
	float_point fUpSelfKernelValue = 0;
	fUpSelfKernelValue = hessianDiag[IdofInstanceOne];

	//for selecting the second instance
	float_point fMinValue;
	fMinValue = hostBuffer[1];
	upValue = -fMinValue;

	//get block level min (-b_ij*b_ij/a_ij)
	GetBlockMinLowValue<<<gridSize, BLOCK_SIZE>>>
						   (devYiGValue, devAlpha, devLabel, CforNegative, numTrainingInstance, devHessianDiag,
							devHessianInstanceRow1, upValue, fUpSelfKernelValue, devBlockMin, devBlockMinGlobalKey,
							devBlockMinYiGValue);

	//get global min
	GetGlobalMin<<<1, BLOCK_SIZE>>>
					(devBlockMin, devBlockMinGlobalKey,
					 numOfBlock, devYiGValue, devHessianInstanceRow1, devBuffer);

	//get global min YiFValue
	//0 is the size of dynamically allocated shared memory inside kernel
	GetGlobalMin<<<1, BLOCK_SIZE>>>(devBlockMinYiGValue, numOfBlock, devBuffer);

	//copy result back to host
	hipMemcpy(hostBuffer, devBuffer, sizeof(float_point) * 4, hipMemcpyDeviceToHost);
    TIMER_STOP(selectTimer)
}

/**
 * @brief: update two weights
 */
void BaseSMO::UpdateTwoWeight(float_point fMinLowValue, float_point fMinValue, int nHessianRowOneInMatrix,
                                     int nHessianRowTwoInMatrix, float_point fKernelValue, float_point &fY1AlphaDiff,
                                     float_point &fY2AlphaDiff, const int *label, float_point C) {
    //get YiGValue for sample one and two
    float_point fAlpha2 = 0;
    float_point fYiFValue2 = 0;
    fAlpha2 = alpha[IdofInstanceTwo];	//reserved for svm regression
    fYiFValue2 = fMinLowValue;

    //get alpha values of sample
    float_point fAlpha1 = 0;
    float_point fYiFValue1 = 0;
    fAlpha1 = alpha[IdofInstanceOne];	//reserved for svm regression
    fYiFValue1 = fMinValue;

    //Get K(x_up, x_up), and K(x_low, x_low)
    float_point fDiag1 = 0, fDiag2 = 0;
    fDiag1 = hessianDiag[nHessianRowOneInMatrix];
    fDiag2 = hessianDiag[nHessianRowTwoInMatrix];

    //get labels of sample one and two
    int nLabel1 = 0, nLabel2 = 0;
    nLabel1 = label[IdofInstanceOne];
    nLabel2 = label[IdofInstanceTwo];

    //compute eta
    float_point eta = fDiag1 + fDiag2 - 2 * fKernelValue;
    if (eta <= 0)
        eta = TAU;

    float_point fCost1, fCost2;
//	fCost1 = Get_C(nLabel1);
//	fCost2 = Get_C(nLabel2);
    fCost1 = fCost2 = C;

    //keep old yi*alphas
    fY1AlphaDiff = nLabel1 * fAlpha1;
    fY2AlphaDiff = nLabel2 * fAlpha2;

    //get new alpha values
    int nSign = nLabel2 * nLabel1;
    if (nSign < 0) {
        float_point fDelta = (-nLabel1 * fYiFValue1 - nLabel2 * fYiFValue2) / eta; //(-fYiFValue1 - fYiFValue2) / eta;
        float_point fAlphaDiff = fAlpha1 - fAlpha2;
        fAlpha1 += fDelta;
        fAlpha2 += fDelta;

        if (fAlphaDiff > 0) {
            if (fAlpha2 < 0) {
                fAlpha2 = 0;
                fAlpha1 = fAlphaDiff;
            }
        } else {
            if (fAlpha1 < 0) {
                fAlpha1 = 0;
                fAlpha2 = -fAlphaDiff;
            }
        }

        if (fAlphaDiff > fCost1 - fCost2) {
            if (fAlpha1 > fCost1) {
                fAlpha1 = fCost1;
                fAlpha2 = fCost1 - fAlphaDiff;
            }
        } else {
            if (fAlpha2 > fCost2) {
                fAlpha2 = fCost2;
                fAlpha1 = fCost2 + fAlphaDiff;
            }
        }
    } //end if nSign < 0
    else {
        float_point fDelta = (nLabel1 * fYiFValue1 - nLabel2 * fYiFValue2) / eta;
        float_point fSum = fAlpha1 + fAlpha2;
        fAlpha1 -= fDelta;
        fAlpha2 += fDelta;

        if (fSum > fCost1) {
            if (fAlpha1 > fCost1) {
                fAlpha1 = fCost1;
                fAlpha2 = fSum - fCost1;
            }
        } else {
            if (fAlpha2 < 0) {
                fAlpha2 = 0;
                fAlpha1 = fSum;
            }
        }
        if (fSum > fCost2) {
            if (fAlpha2 > fCost2) {
                fAlpha2 = fCost2;
                fAlpha1 = fSum - fCost2;
            }
        } else {
            if (fAlpha1 < 0) {
                fAlpha1 = 0;
                fAlpha2 = fSum;
            }
        }
    }//end get new alpha values

    alpha[IdofInstanceOne] = fAlpha1;
    alpha[IdofInstanceTwo] = fAlpha2;

    //get alpha difference
    fY1AlphaDiff = nLabel1 * fAlpha1 - fY1AlphaDiff; //(alpha1' - alpha1) * y1
    fY2AlphaDiff = nLabel2 * fAlpha2 - fY2AlphaDiff;
}

/*
 * @brief: update the optimality indicator
 */
void BaseSMO::UpdateYiGValue(int numTrainingInstance, float_point fY1AlphaDiff, float_point fY2AlphaDiff)
{
    float_point fAlpha1 = alpha[IdofInstanceOne];
    float_point fAlpha2 = alpha[IdofInstanceTwo];
    //update yiFvalue
    //copy new alpha values to device
    hostBuffer[0] = IdofInstanceOne;
    hostBuffer[1] = fAlpha1;
    hostBuffer[2] = IdofInstanceTwo;
    hostBuffer[3] = fAlpha2;
    checkCudaErrors(hipMemcpy(devBuffer, hostBuffer, sizeof(float_point) * 4, hipMemcpyHostToDevice));
    UpdateYiFValueKernel <<< gridSize, BLOCK_SIZE >>> (devAlpha, devBuffer, devYiGValue,
            devHessianInstanceRow1, devHessianInstanceRow2,
            fY1AlphaDiff, fY2AlphaDiff, numTrainingInstance);
    hipDeviceSynchronize();
}

void BaseSMO::configureCudaKernel(int numOfTrainingInstance) {
    //configure cuda kernel
    numOfBlock = Ceil(numOfTrainingInstance, BLOCK_SIZE);
    gridSize = dim3(numOfBlock > NUM_OF_BLOCK ? NUM_OF_BLOCK : numOfBlock, Ceil(numOfBlock, NUM_OF_BLOCK));
}

